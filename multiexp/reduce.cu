#include "hip/hip_runtime.h"
#include <cstdint>
#include <climits>
#include <utility>

#include "curves.cu"
#include "utils.cu"

template<int C, int L = 753, int B = 256>
struct ec_multiexp_config {
    static constexpr int THREADS_PER_BLOCK = B;
    static_assert(THREADS_PER_BLOCK % HIPCUB_WARP_THREADS == 0, "block size must be multiple of warp size");
    static constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / HIPCUB_WARP_THREADS;
    static constexpr int ELTS_PER_BLOCK = WARPS_PER_BLOCK * ELTS_PER_WARP;

    static constexpr int WIN_BITS = C;
    static constexpr int NWIN = (L + WIN_BITS - 1) / WIN_BITS;
    static constexpr int WIN_MASK = (1 << WIN_BITS) - 1;
    static constexpr int BUCKETS_PER_WIN = 1 << WIN_BITS;

    static constexpr int BUCKET_SZ_LEN = BUCKETS_PER_WIN * NWIN;
    static constexpr int BUCKET_MAP_LEN = (BUCKETS_PER_WIN - 1) * NWIN;
    static constexpr int BUCKET_SZ_OFF = 0;
    static constexpr int BUCKET_MAP_OFF = BUCKET_SZ_OFF + BUCKET_SZ_LEN;
    static constexpr int BUCKET_IDX_OFF = BUCKET_MAP_OFF + BUCKET_MAP_LEN;
};

__device__ int
get_idx() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int warp_id = idx >> CUB_PTX_LOG_WARP_THREADS, lane_id = idx & (HIPCUB_WARP_THREADS - 1);
    int lane_elt_idx = lane_id / ELT_LIMBS;
    if (lane_elt_idx >= ELTS_PER_WARP) {
        return INT_MAX;
    }
    return warp_id * ELTS_PER_WARP + lane_elt_idx;
}

template<typename T>
struct dev_cmp {
    __device__ bool operator()(T x, T y) {
        return x < y;
    }
};

template<typename Fr>
__global__ void
ec_scalar_from_monty(var *scalars_, size_t N) {
    int idx = get_idx();
    if (idx >= N) {
        return;
    }
    var *p = scalars_ + idx * ELT_LIMBS;
    Fr x;
    Fr::load(x, p);
    Fr::from_monty(x, x);
    Fr::store(p, x);
}

// extract bucket id for every window of every coefficient
template<typename C>
__global__ void
ec_multiexp_scan_idx(const var *scalars, int *out_keys, int *out_items, size_t n) {
    // we assume n * NWIN < INT_MAX
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = idx / C::NWIN;
    int win_id = idx % C::NWIN;
    if (i >= n) {
        return;
    }
    const var *scalar = scalars + i * ELT_LIMBS;
    int win_off = win_id * C::WIN_BITS;
    int j = win_off / digit::BITS, k = win_off % digit::BITS;
    int bucket = (scalar[j] >> k) & C::WIN_MASK;
    if (digit::BITS - k < C::WIN_BITS && j < ELT_LIMBS - 1) {
        bucket |= (scalar[j + 1] << (digit::BITS - k)) & C::WIN_MASK;
    }

    out_keys[idx] = (win_id << C::WIN_BITS) | bucket;
    out_items[idx] = i;
}

// find sizes of the buckets by differentiation
template<typename C>
__global__ void
ec_multiexp_scan_sz(const int *keys, size_t n, int *out) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int win = idx / (n + 1);
    int i = idx % (n + 1);
    if (win >= C::NWIN) {
        return;
    }
    int lb = i == 0 ? 0 : keys[i - 1 + win * n] & C::WIN_MASK;
    int rb = i == n ? C::BUCKETS_PER_WIN : keys[i + win * n] & C::WIN_MASK;

    // use loop to handle empty buckets
    for (int j = lb; j < rb; j++) {
        out[(win << C::WIN_BITS) | j] = i + win * n;
    }
}

template<typename C>
__global__ void
ec_multiexp_balance(int *bucket_sz, int *bucket_map, float target) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= C::NWIN) {
        return;
    }
    auto cur_sz = bucket_sz + idx * C::BUCKETS_PER_WIN;
    auto cur_map = bucket_map + idx * (C::BUCKETS_PER_WIN - 1);
    int empty_cnt = 0;
    for (int i = 0; i < C::BUCKETS_PER_WIN - 1; i++) {
        if (cur_sz[i] == cur_sz[i + 1]) {
            empty_cnt++;
        }
    }
    int buf[C::BUCKETS_PER_WIN] = { cur_sz[0] }, cnt = 1;
    for (int i = 0; i < C::BUCKETS_PER_WIN - 1; i++) {
        cur_map[i] = cnt - 1;
        int d = cur_sz[i + 1] - cur_sz[i];
        if (d == 0) {
            continue;
        }
        int k = lroundf(d / target);
        k = CUB_MIN(k, empty_cnt + 1);
        if (k <= 1) {
            buf[cnt++] = cur_sz[i + 1];
            continue;
        }
        int off = 0;
        for (int j = 0; j < k; j++) {
            off += d / k + (d % k > j);
            buf[cnt + j] = cur_sz[i] + off;
        }
        cnt += k;
        empty_cnt -= k - 1;
    }
    for (; cnt < C::BUCKETS_PER_WIN; cnt++) {
        buf[cnt] = buf[cnt - 1];
    }
    memcpy(cur_sz, buf, C::BUCKETS_PER_WIN * sizeof(int));
}

template<typename C>
void ec_multiexp_scan_mem_size(size_t n, size_t *temp_size, size_t *out_size) {
    size_t sort_size;
    HipcubDebug(hipcub::DeviceMergeSort::SortPairs(nullptr, sort_size, (int *)nullptr, (int *)nullptr, n * C::NWIN, dev_cmp<int>()));
    *temp_size = n * C::NWIN * sizeof(int) + sort_size; // keys + cub temp
    *out_size = (C::BUCKET_SZ_LEN + C::BUCKET_MAP_LEN + n * C::NWIN) * sizeof(int); // sz + map + idx
}

// put coefficient indices into buckets
// input is modified for conversion from montegomery form
template<typename Fr, typename C>
void
ec_multiexp_scan(var *scalars, int *out, size_t n, void *temp, size_t temp_size, hipStream_t stream) {
    auto idx_size = n * C::NWIN;
    auto out_sz = out + C::BUCKET_SZ_OFF;
    auto out_map = out + C::BUCKET_MAP_OFF;
    auto out_idx = out + C::BUCKET_IDX_OFF;
    auto keys = (int *)temp;
    auto sort_temp = (void *)(keys + idx_size);
    auto sort_temp_size = temp_size - idx_size * sizeof(int);

    // convert from montegomery form
    int nblocks = (n + C::ELTS_PER_BLOCK - 1) / C::ELTS_PER_BLOCK;
    ec_scalar_from_monty<Fr><<<nblocks, C::THREADS_PER_BLOCK, 0, stream>>>(scalars, n);
    HipcubDebug(hipGetLastError());

    // extract bucket id
    nblocks = (idx_size + C::THREADS_PER_BLOCK - 1) / C::THREADS_PER_BLOCK;
    ec_multiexp_scan_idx<C><<<nblocks, C::THREADS_PER_BLOCK, 0, stream>>>(scalars, keys, out_idx, n);
    HipcubDebug(hipGetLastError());

    // sort so that items with same window & id are grouped together
    HipcubDebug(hipcub::DeviceMergeSort::SortPairs(sort_temp, sort_temp_size, keys, out_idx, idx_size, dev_cmp<int>(), stream));

    // find bucket sizes
    nblocks = ((n + 1) * C::NWIN + C::THREADS_PER_BLOCK - 1) / C::THREADS_PER_BLOCK;
    ec_multiexp_scan_sz<C><<<nblocks, C::THREADS_PER_BLOCK, 0, stream>>>(keys, n, out_sz);
    HipcubDebug(hipGetLastError());

    // attempt to balance bucket sizes (mainly for last window)
    nblocks = (C::NWIN + C::THREADS_PER_BLOCK - 1) / C::THREADS_PER_BLOCK;
    ec_multiexp_balance<C><<<nblocks, C::THREADS_PER_BLOCK, 0, stream>>>(out_sz, out_map, (float)n / C::BUCKETS_PER_WIN);
    HipcubDebug(hipGetLastError());
}

// TODO: use whole warp for a bucket?
// calculate sum in each bucket
template<typename EC, typename C>
__global__ void
ec_multiexp_point_merge(const var *pts, const int *bucket_info, var *out, size_t off, size_t n) {
    int idx = get_idx();
    int group = idx % ELTS_PER_WARP;
    int warp = idx / ELTS_PER_WARP;
    int win = warp / (C::BUCKETS_PER_WIN - 1);
    int bucket = warp % (C::BUCKETS_PER_WIN - 1);
    if (win >= C::NWIN) {
        return;
    }
    const int *bucket_sz = bucket_info + C::BUCKET_SZ_OFF, *bucket_idx = bucket_info + C::BUCKET_IDX_OFF;
    EC x, m;
    EC::set_zero(x);
    // bucket 0 is skipped
    int l = bucket_sz[bucket + win * C::BUCKETS_PER_WIN], r = bucket_sz[bucket + 1 + win * C::BUCKETS_PER_WIN];
    for (int j = l + group; j < r; j += ELTS_PER_WARP) {
        int k = bucket_idx[j] - off;
        if (k >= 0 && k < n) {
            EC::load_affine(m, pts + k * EC::NLIMBS_AFF);
            EC::mixed_add(x, x, m);
        }
    }
    EC::store_jac(out + idx * EC::NLIMBS, x);
}

// reduce all buckets in a window
template<typename EC, typename C>
__global__ void
ec_multiexp_bucket_reduce(var *buckets, const int *bucket_map, var *out) {
    int idx = get_idx();
    if (idx >= C::NWIN) {
        return;
    }

    auto cur_buckets = buckets + idx * (C::BUCKETS_PER_WIN - 1) * ELTS_PER_WARP * EC::NLIMBS;
    auto cur_map = bucket_map + idx * (C::BUCKETS_PER_WIN - 1);

    EC x, y;
    for (int i = (C::BUCKETS_PER_WIN - 1) * ELTS_PER_WARP - 2; i >= 0; i--) {
        EC::load_jac(x, cur_buckets + i * EC::NLIMBS);
        EC::load_jac(y, cur_buckets + (i + 1) * EC::NLIMBS);
        EC::add(x, x, y);
        EC::store_jac(cur_buckets + i * EC::NLIMBS, x);
    }

    EC::set_zero(x);
    for (int i = C::BUCKETS_PER_WIN - 2; i >= 0; i--) {
        int k = cur_map[i];
        if (k < C::BUCKETS_PER_WIN - 1) {
            EC::load_jac(y, cur_buckets + k * ELTS_PER_WARP * EC::NLIMBS);
            EC::add(x, x, y);
        }
    }

    // TODO: hard to parallelize, consider preprocessing
    for (int i = 0; i < idx * C::WIN_BITS; i++) {
        EC::dbl(x, x);
    }

    EC::store_jac(out + idx * EC::NLIMBS, x);
}

template< typename EC >
__global__ void
ec_sum_all(var *X, const var *Y, size_t n)
{
    int idx = get_idx();

    if (idx < n) {
        EC z, x, y;
        int off = idx * EC::NELTS * ELT_LIMBS;

        EC::load_jac(x, X + off);
        EC::load_jac(y, Y + off);

        EC::add(z, x, y);

        EC::store_jac(X + off, z);
    }
}

template<typename EC, typename C>
void ec_multiexp_pippenger_mem_size(size_t *temp_size, size_t *out_size) {
    *temp_size = (C::BUCKETS_PER_WIN - 1) * C::NWIN * ELTS_PER_WARP * EC::NLIMBS * sizeof(var); // output of point merge
    *out_size = C::NWIN * EC::NLIMBS * sizeof(var); // avoid extra copy after inplace sum
}

template<typename EC, typename C>
void
ec_multiexp_pippenger(const var *pts, const int *bucket_info, var *out, void *temp, size_t off, size_t n, hipStream_t stream) {
    int nblocks = ((C::BUCKETS_PER_WIN - 1) * C::NWIN + C::WARPS_PER_BLOCK - 1) / C::WARPS_PER_BLOCK;
    ec_multiexp_point_merge<EC, C><<<nblocks, C::THREADS_PER_BLOCK, 0, stream>>>(pts, bucket_info, (var *)temp, off, n);
    HipcubDebug(hipGetLastError());

    auto bucket_map = bucket_info + C::BUCKET_MAP_OFF;
    nblocks = (C::NWIN + C::ELTS_PER_BLOCK - 1) / C::ELTS_PER_BLOCK;
    ec_multiexp_bucket_reduce<EC, C><<<nblocks, C::THREADS_PER_BLOCK, 0, stream>>>((var *)temp, bucket_map, out);
    HipcubDebug(hipGetLastError());

    for (size_t r = C::NWIN & 1, m = C::NWIN / 2; m != 0; r = m & 1, m >>= 1) {
        nblocks = (m + C::ELTS_PER_BLOCK - 1) / C::ELTS_PER_BLOCK;
        ec_sum_all<EC><<<nblocks, C::THREADS_PER_BLOCK, 0, stream>>>(out, out + m * EC::NLIMBS, m);
        if (r) {
            ec_sum_all<EC><<<1, C::THREADS_PER_BLOCK, 0, stream>>>(out, out + 2 * m * EC::NLIMBS, 1);
        }
    }
    HipcubDebug(hipGetLastError());
}
