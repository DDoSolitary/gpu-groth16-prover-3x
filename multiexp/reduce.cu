#include "hip/hip_runtime.h"
#include <cstdint>
#include <vector>
#include <chrono>
#include <memory>
#include <algorithm>
#include <cstdint>

#include "curves.cu"

template<typename Fr>
__global__ void
ec_scalar_from_monty_kernel(var *scalars_, size_t N) {
    int T = threadIdx.x, B = blockIdx.x, D = blockDim.x;
    int elts_per_block = D / BIG_WIDTH;
    int tileIdx = T / BIG_WIDTH;

    int idx = elts_per_block * B + tileIdx;
    if (idx >= N) {
        return;
    }
    var *p = scalars_ + idx * ELT_LIMBS;
    Fr x;
    Fr::load(x, p);
    Fr::from_monty(x, x);
    Fr::store(p, x);
}

// C is the size of the precomputation
// R is the number of points we're handling per thread
template< typename EC, int C = 4, int RR = 8 >
__global__ void
ec_multiexp_straus(var *out, const var *multiples_, const var *scalars_, size_t N)
{
    int T = threadIdx.x, B = blockIdx.x, D = blockDim.x;
    int elts_per_block = D / BIG_WIDTH;
    int tileIdx = T / BIG_WIDTH;

    int idx = elts_per_block * B + tileIdx;

    size_t n = (N + RR - 1) / RR;
    if (idx < n) {
        // TODO: Treat remainder separately so R can remain a compile time constant
        size_t R = (idx < n - 1) ? RR : (N % RR);

        typedef typename EC::group_type Fr;
        static constexpr int JAC_POINT_LIMBS = 3 * EC::field_type::DEGREE * ELT_LIMBS;
        static constexpr int AFF_POINT_LIMBS = 2 * EC::field_type::DEGREE * ELT_LIMBS;
        int out_off = idx * JAC_POINT_LIMBS;
        int m_off = idx * RR * AFF_POINT_LIMBS;
        int s_off = idx * RR * ELT_LIMBS;

        const var *multiples = multiples_ + m_off;
        // TODO: Consider loading multiples and/or scalars into shared memory

        // i is smallest multiple of C such that i > 753
        int i = C * ((753 + C - 1) / C); // C * ceiling(753/C)
        assert((i - C * 753) < C);
        static constexpr var C_MASK = (1U << C) - 1U;

        EC x;
        EC::set_zero(x);
        while (i >= C) {
            EC::template mul_2exp<C>(x, x);
            i -= C;

            int q = i / digit::BITS, r = i % digit::BITS;
            for (int j = 0; j < R; ++j) {
                auto scalar = scalars_ + s_off + j * ELT_LIMBS;
                var s = scalar[q];
                var win = (s >> r) & C_MASK;
                // Handle case where C doesn't divide digit::BITS
                int bottom_bits = digit::BITS - r;
                // detect when window overlaps digit boundary
                if (bottom_bits < C) {
                    s = scalar[q + 1];
                    win |= (s << bottom_bits) & C_MASK;
                }
                if (win > 0) {
                    EC m;
                    //EC::add(x, x, multiples[win - 1][j]);
                    EC::load_affine(m, multiples + ((win-1)*N + j)*AFF_POINT_LIMBS);
                    EC::mixed_add(x, x, m);
                }
            }
        }
        EC::store_jac(out + out_off, x);
    }
}

template< typename EC >
__global__ void
ec_multiexp(var *X, const var *W, size_t n)
{
    int T = threadIdx.x, B = blockIdx.x, D = blockDim.x;
    int elts_per_block = D / BIG_WIDTH;
    int tileIdx = T / BIG_WIDTH;

    int idx = elts_per_block * B + tileIdx;

    if (idx < n) {
        typedef typename EC::group_type Fr;
        EC x;
        Fr w;
        int x_off = idx * EC::NELTS * ELT_LIMBS;
        int w_off = idx * ELT_LIMBS;

        EC::load_affine(x, X + x_off);
        Fr::load(w, W + w_off);

        // We're given W in Monty form for some reason, so undo that.
        Fr::from_monty(w, w);
        EC::mul(x, w.a, x);

        EC::store_jac(X + x_off, x);
    }
}

template< typename EC >
__global__ void
ec_sum_all(var *X, const var *Y, size_t n)
{
    int T = threadIdx.x, B = blockIdx.x, D = blockDim.x;
    int elts_per_block = D / BIG_WIDTH;
    int tileIdx = T / BIG_WIDTH;

    int idx = elts_per_block * B + tileIdx;

    if (idx < n) {
        EC z, x, y;
        int off = idx * EC::NELTS * ELT_LIMBS;

        EC::load_jac(x, X + off);
        EC::load_jac(y, Y + off);

        EC::add(z, x, y);

        EC::store_jac(X + off, z);
    }
}

static constexpr size_t threads_per_block = 256;

template<typename EC>
void
ec_scalar_from_monty(var *scalars, size_t N) {
    size_t nblocks = (N * BIG_WIDTH + threads_per_block - 1) / threads_per_block;
    ec_scalar_from_monty_kernel<typename EC::group_type><<<nblocks, threads_per_block>>>(scalars, N);
}

template< typename EC, int C, int R >
void
ec_reduce_straus(hipStream_t &strm, var *out, const var *multiples, const var *scalars, size_t N)
{
    hipStreamCreate(&strm);

    static constexpr size_t pt_limbs = EC::NELTS * ELT_LIMBS;
    size_t n = (N + R - 1) / R;

    size_t nblocks = (n * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

    ec_multiexp_straus<EC, C, R><<< nblocks, threads_per_block, 0, strm>>>(out, multiples, scalars, N);

    size_t r = n & 1, m = n / 2;
    for ( ; m != 0; r = m & 1, m >>= 1) {
        nblocks = (m * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

        ec_sum_all<EC><<<nblocks, threads_per_block, 0, strm>>>(out, out + m*pt_limbs, m);
        if (r)
            ec_sum_all<EC><<<1, threads_per_block, 0, strm>>>(out, out + 2*m*pt_limbs, 1);
    }
}

template< typename EC >
void
ec_reduce(hipStream_t &strm, var *X, const var *w, size_t n)
{
    hipStreamCreate(&strm);

    size_t nblocks = (n * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

    // FIXME: Only works on Pascal and later.
    //auto grid = cg::this_grid();
    ec_multiexp<EC><<< nblocks, threads_per_block, 0, strm>>>(X, w, n);

    static constexpr size_t pt_limbs = EC::NELTS * ELT_LIMBS;

    size_t r = n & 1, m = n / 2;
    for ( ; m != 0; r = m & 1, m >>= 1) {
        nblocks = (m * BIG_WIDTH + threads_per_block - 1) / threads_per_block;

        ec_sum_all<EC><<<nblocks, threads_per_block, 0, strm>>>(X, X + m*pt_limbs, m);
        if (r)
            ec_sum_all<EC><<<1, threads_per_block, 0, strm>>>(X, X + 2*m*pt_limbs, 1);
        // TODO: Not sure this is really necessary.
        //grid.sync();
    }
}

static inline double as_mebibytes(size_t n) {
    return n / (long double)(1UL << 20);
}

void print_meminfo(size_t allocated) {
    size_t free_mem, dev_mem;
    hipMemGetInfo(&free_mem, &dev_mem);
    fprintf(stderr, "Allocated %zu bytes; device has %.1f MiB free (%.1f%%).\n",
            allocated,
            as_mebibytes(free_mem),
            100.0 * free_mem / dev_mem);
}

struct CudaFree {
    void operator()(var *mem) { hipFree(mem); }
};
typedef std::unique_ptr<var, CudaFree> var_ptr;

var_ptr
allocate_memory(size_t nbytes, int dbg = 0) {
    var *mem = nullptr;
    hipMalloc(&mem, nbytes);
    if (mem == nullptr) {
        fprintf(stderr, "Failed to allocate enough device memory\n");
        abort();
    }
    if (dbg)
        print_meminfo(nbytes);
    return var_ptr(mem);
}

struct CudaFreeHost {
    void operator()(var *mem) { hipHostFree(mem); }
};

std::unique_ptr<var, CudaFreeHost>
allocate_host_memory(size_t nbytes) {
    var *mem = nullptr;
    hipHostAlloc(&mem, nbytes, hipHostMallocDefault);
    if (mem == nullptr) {
        fprintf(stderr, "Failed to allocate enough host memory\n");
        abort();
    }
    return std::unique_ptr<var, CudaFreeHost>(mem);
}

var_ptr read_file_chunked(FILE *f, size_t n) {
    auto dev_buf = allocate_memory(n);
    auto dev_ptr = (char *)dev_buf.get();
    auto bufsz = std::min(n, (size_t)INT32_MAX); // Iluvatar's hipMemcpy fails when size is above this
    auto host_buf = allocate_host_memory(bufsz);
    auto host_ptr = (char *)host_buf.get();
    for (size_t off = 0; off < n; off += bufsz) {
        auto sz = std::min(n, off + bufsz) - off;
        if (fread(host_ptr, sz, 1, f) < 1) {
            fprintf(stderr, "Failed to read input\n");
            abort();
        }
        hipMemcpy(dev_ptr + off, host_ptr, sz, hipMemcpyHostToDevice);
    }
    return dev_buf;
}

var_ptr
load_scalars(size_t n, FILE *inputs)
{
    static constexpr size_t scalar_bytes = ELT_BYTES;
    size_t total_bytes = n * scalar_bytes;

    return read_file_chunked(inputs, total_bytes);
}

template< typename EC >
var_ptr
load_points_affine(size_t n, FILE *inputs)
{
    typedef typename EC::field_type FF;

    static constexpr size_t coord_bytes = FF::DEGREE * ELT_BYTES;
    static constexpr size_t aff_pt_bytes = 2 * coord_bytes;

    size_t total_aff_bytes = n * aff_pt_bytes;

    return read_file_chunked(inputs, total_aff_bytes);
}
