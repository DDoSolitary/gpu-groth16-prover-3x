#include <string>
#include <chrono>
#include <thread>
#include <map>
#include <functional>

#define NDEBUG 1
#define HIPCUB_STDERR

#include <prover_reference_functions.hpp>

#include "multiexp/reduce.cu"

// This is where all the FFTs happen

// template over the bundle of types and functions.
// Overwrites ca!
template <typename B>
typename B::vector_Fr *compute_H(size_t d, typename B::vector_Fr *ca,
                                 typename B::vector_Fr *cb,
                                 typename B::vector_Fr *cc) {
  auto domain = B::get_evaluation_domain(d + 1);

  B::domain_iFFT(domain, ca);
  B::domain_iFFT(domain, cb);

  B::domain_cosetFFT(domain, ca);
  B::domain_cosetFFT(domain, cb);

  // Use ca to store H
  auto H_tmp = ca;

  size_t m = B::domain_get_m(domain);
  // for i in 0 to m: H_tmp[i] *= cb[i]
  B::vector_Fr_muleq(H_tmp, cb, m);

  B::domain_iFFT(domain, cc);
  B::domain_cosetFFT(domain, cc);

  m = B::domain_get_m(domain);

  // for i in 0 to m: H_tmp[i] -= cc[i]
  B::vector_Fr_subeq(H_tmp, cc, m);

  B::domain_divide_by_Z_on_coset(domain, H_tmp);

  B::domain_icosetFFT(domain, H_tmp);

  m = B::domain_get_m(domain);
  typename B::vector_Fr *H_res = B::vector_Fr_zeros(m + 1);
  B::vector_Fr_copy_into(H_tmp, H_res, m);
  return H_res;
}

static size_t read_size_t(FILE* input) {
  size_t n;
  fread((void *) &n, sizeof(size_t), 1, input);
  return n;
}

template< typename B >
struct ec_type;

template<>
struct ec_type<mnt4753_libsnark> {
    typedef ECp_MNT4 ECp;
    typedef ECp2_MNT4 ECpe;
};

template<>
struct ec_type<mnt6753_libsnark> {
    typedef ECp_MNT6 ECp;
    typedef ECp3_MNT6 ECpe;
};


void
check_trailing(FILE *f, const char *name) {
    long bytes_remaining = 0;
    while (fgetc(f) != EOF)
        ++bytes_remaining;
    if (bytes_remaining > 0)
        fprintf(stderr, "!! Trailing characters in \"%s\": %ld\n", name, bytes_remaining);
}


static inline auto now() -> decltype(std::chrono::high_resolution_clock::now()) {
    return std::chrono::high_resolution_clock::now();
}

template<typename T>
void
print_time(T &t1, const char *str) {
    auto t2 = std::chrono::high_resolution_clock::now();
    auto tim = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    printf("%s: %ld ms\n", str, tim);
    t1 = t2;
}

template<typename B, typename C>
void run_prover(
        const char *params_path,
        const char *input_path,
        const char *output_path)
{
    typedef typename ec_type<B>::ECp ECp;
    typedef typename ec_type<B>::ECpe ECpe;

    typedef typename B::G1 G1;
    typedef typename B::G2 G2;

    hipStream_t sA, sB1, sB2, sL;
    HipcubDebug(hipStreamCreate(&sA));
    HipcubDebug(hipStreamCreate(&sB1));
    HipcubDebug(hipStreamCreate(&sB2));
    HipcubDebug(hipStreamCreate(&sL));

    B::init_public_params();

    auto beginning = now();
    auto t = beginning;

    FILE *params_file = fopen(params_path, "r");

    size_t d = read_size_t(params_file);
    size_t m = read_size_t(params_file);
    printf("d = %zu, m = %zu\n", d, m);

    size_t A_pts_size = ECp::NBYTES_AFF * (m + 1);
    auto A_pts = allocate_memory(A_pts_size);
    auto A_pts_h = allocate_host_memory(A_pts_size);
    size_t B1_pts_size = ECp::NBYTES_AFF * (m + 1);
    auto B1_pts = allocate_memory(B1_pts_size);
    auto B1_pts_h = allocate_host_memory(B1_pts_size);
    size_t B2_pts_size = ECpe::NBYTES_AFF * (m + 1);
    auto B2_pts = allocate_memory(B2_pts_size);
    auto B2_pts_h = allocate_host_memory(B2_pts_size);
    size_t L_pts_size = ECp::NBYTES_AFF * (m - 1);
    auto L_pts = allocate_memory(L_pts_size);
    auto L_pts_h = allocate_host_memory(L_pts_size);
    size_t H_pts_size = ECp::NBYTES_AFF * d;
    auto H_pts_h = allocate_host_memory(H_pts_size);

    auto input_w_size = (m + 1) * ELT_BYTES;
    auto input_others_size = ((d + 1) * 3 + 1) * ELT_BYTES;
    auto inputs_h = allocate_host_memory<char>(input_w_size + input_others_size);
    auto w = allocate_memory(input_w_size);

    size_t scan_temp_size, scan_out_size;
    ec_multiexp_scan_mem_size<C>(m + 1, &scan_temp_size, &scan_out_size);
    size_t temp_size_G1, out_size_G1;
    ec_multiexp_pippenger_mem_size<ECp, C>(&temp_size_G1, &out_size_G1);
    size_t temp_size_G2, out_size_G2;
    ec_multiexp_pippenger_mem_size<ECpe, C>(&temp_size_G2, &out_size_G2);

    auto scan_temp = allocate_memory<void>(scan_temp_size);
    auto scan_out = allocate_memory<int>(scan_out_size);
    auto temp_A = allocate_memory<void>(temp_size_G1);
    auto out_A = allocate_memory<var>(out_size_G1);
    auto temp_B1 = allocate_memory<void>(temp_size_G1);
    auto out_B1 = allocate_memory<var>(out_size_G1);
    auto temp_B2 = allocate_memory<void>(temp_size_G2);
    auto out_B2 = allocate_memory<var>(out_size_G2);
    auto temp_L = allocate_memory<void>(temp_size_G1);
    auto out_L = allocate_memory<var>(out_size_G1);

    auto out_A_h = allocate_host_memory(ECp::NBYTES);
    auto out_B1_h = allocate_host_memory(ECp::NBYTES);
    auto out_B2_h = allocate_host_memory(ECpe::NBYTES);
    auto out_L_h = allocate_host_memory(ECp::NBYTES);

    print_time(t, "alloc device mem");

    auto params = B::alloc_params(d, m);
    fread(A_pts_h.get(), A_pts_size, 1, params_file);
    HipcubDebug(hipMemcpyAsync(A_pts.get(), A_pts_h.get(), A_pts_size, hipMemcpyHostToDevice));
    // B::read_params_A(params, A_pts_h.get());
    fread(B1_pts_h.get(), B1_pts_size, 1, params_file);
    HipcubDebug(hipMemcpyAsync(B1_pts.get(), B1_pts_h.get(), B1_pts_size, hipMemcpyHostToDevice));
    B::read_params_B1(params, B1_pts_h.get());
    fread(B2_pts_h.get(), B2_pts_size, 1, params_file);
    HipcubDebug(hipMemcpyAsync(B2_pts.get(), B2_pts_h.get(), B2_pts_size, hipMemcpyHostToDevice));
    // B::read_params_B2(params, B2_pts_h.get());
    fread(L_pts_h.get(), L_pts_size, 1, params_file);
    HipcubDebug(hipMemcpyAsync(L_pts.get(), L_pts_h.get(), L_pts_size, hipMemcpyHostToDevice));
    // B::read_params_L(params, L_pts_h.get());
    fread(H_pts_h.get(), H_pts_size, 1, params_file);
    B::read_params_H(params, H_pts_h.get());
    fclose(params_file);
    print_time(t, "load params");

    auto t_main = t;

    FILE *inputs_file = fopen(input_path, "r");
    fread(inputs_h.get(), input_w_size, 1, inputs_file);
    HipcubDebug(hipMemcpyAsync(w.get(), inputs_h.get(), input_w_size, hipMemcpyHostToDevice));
    fread(inputs_h.get() + input_w_size, input_others_size, 1, inputs_file);
    fclose(inputs_file);
    auto inputs = B::make_input(inputs_h.get(), d, m);
    print_time(t, "load inputs");

    auto t_gpu = t;

    ec_multiexp_scan<typename ECp::group_type, C>(w.get(), scan_out.get(), m + 1, scan_temp.get(), scan_temp_size, nullptr);
    ec_multiexp_pippenger<ECp, C>(A_pts.get(), scan_out.get(), out_A.get(), temp_A.get(), 0, m + 1, sA);
    ec_multiexp_pippenger<ECp, C>(B1_pts.get(), scan_out.get(), out_B1.get(), temp_B1.get(), 0, m + 1, sB1);
    ec_multiexp_pippenger<ECpe, C>(B2_pts.get(), scan_out.get(), out_B2.get(), temp_B2.get(), 0, m + 1, sB2);
    ec_multiexp_pippenger<ECp, C>(L_pts.get(), scan_out.get(), out_L.get(), temp_L.get(), 2, m - 1, sL);
    HipcubDebug(hipMemcpyAsync(out_A_h.get(), out_A.get(), ECp::NBYTES, hipMemcpyDeviceToHost, sA));
    HipcubDebug(hipMemcpyAsync(out_B1_h.get(), out_B1.get(), ECp::NBYTES, hipMemcpyDeviceToHost, sB1));
    HipcubDebug(hipMemcpyAsync(out_B2_h.get(), out_B2.get(), ECpe::NBYTES, hipMemcpyDeviceToHost, sB2));
    HipcubDebug(hipMemcpyAsync(out_L_h.get(), out_L.get(), ECp::NBYTES, hipMemcpyDeviceToHost, sL));
    print_time(t_gpu, "gpu launch");

    // Do calculations relating to H on CPU after having set the GPU in
    // motion
    typename B::vector_G1 *H;
    typename B::vector_Fr *coefficients_for_H;
    G1 *evaluation_Ht;
    std::thread cpu1_thread([&]() {
        auto t_cpu1 = now();
        H = B::params_H(params);
        coefficients_for_H =
            compute_H<B>(d, B::input_ca(inputs), B::input_cb(inputs), B::input_cc(inputs));
        print_time(t_cpu1, "cpu fft H");
        evaluation_Ht = B::multiexp_G1(coefficients_for_H, H, d);
        print_time(t_cpu1, "cpu multiexp H");
    });

    HipcubDebug(hipStreamSynchronize(sA));
    G1 *evaluation_At = B::read_pt_ECp(out_A_h.get());
    HipcubDebug(hipStreamSynchronize(sB1));
    G1 *evaluation_Bt1 = B::read_pt_ECp(out_B1_h.get());
    HipcubDebug(hipStreamSynchronize(sB2));
    G2 *evaluation_Bt2 = B::read_pt_ECpe(out_B2_h.get());
    HipcubDebug(hipStreamSynchronize(sL));
    G1 *evaluation_Lt = B::read_pt_ECp(out_L_h.get());
    print_time(t, "gpu e2e");

    cpu1_thread.join();
    print_time(t, "cpu 1 wait");

    auto scaled_Bt1 = B::G1_scale(B::input_r(inputs), evaluation_Bt1);
    auto Lt1_plus_scaled_Bt1 = B::G1_add(evaluation_Lt, scaled_Bt1);
    auto final_C = B::G1_add(evaluation_Ht, Lt1_plus_scaled_Bt1);

    print_time(t, "cpu 2");

    B::groth16_output_write(evaluation_At, evaluation_Bt2, final_C, output_path);

    print_time(t, "store");

    print_time(t_main, "Total time from input to output: ");

    HipcubDebug(hipStreamDestroy(sA));
    HipcubDebug(hipStreamDestroy(sB1));
    HipcubDebug(hipStreamDestroy(sB2));
    HipcubDebug(hipStreamDestroy(sL));

    B::delete_vector_G1(H);

    B::delete_G1(evaluation_At);
    B::delete_G1(evaluation_Bt1);
    B::delete_G2(evaluation_Bt2);
    B::delete_G1(evaluation_Ht);
    B::delete_G1(evaluation_Lt);
    B::delete_G1(scaled_Bt1);
    B::delete_G1(Lt1_plus_scaled_Bt1);
    B::delete_vector_Fr(coefficients_for_H);
    B::delete_groth16_input(inputs);
    B::delete_groth16_params(params);

    print_time(t, "cleanup");
    print_time(beginning, "Total runtime (incl. file reads)");
}

int main(int, char **argv) {
  setbuf(stdout, NULL);
  std::string curve(argv[1]);
  std::string mode(argv[2]);

  const char *params_path = argv[3];

  if (mode == "compute") {
      const char *input_path = argv[4];
      const char *output_path = argv[5];

      using config = ec_multiexp_config<MULTIEXP_WIN_BITS, 753, MULTIEXP_BLOCK_SIZE>;

      if (curve == "MNT4753") {
          run_prover<mnt4753_libsnark, config>(params_path, input_path, output_path);
      } else if (curve == "MNT6753") {
          run_prover<mnt6753_libsnark, config>(params_path, input_path, output_path);
      }
  }

  return 0;
}
